#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"

#define THREADS_PER_BLOCK 256

// helper function to round an integer up to the next power of 2
static inline int nextPow2(int n)
{
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

// exclusive_scan --
//
// Implementation of an exclusive scan on global memory array `input`,
// with results placed in global memory `result`.
//
// N is the logical size of the input and output arrays, however
// students can assume that both the start and result arrays we
// allocated with next power-of-two sizes as described by the comments
// in cudaScan().  This is helpful, since your parallel scan
// will likely write to memory locations beyond N, but of course not
// greater than N rounded up to the next power of 2.
//
// Also, as per the comments in cudaScan(), you can implement an
// "in-place" scan, since the timing harness makes a copy of input and
// places it in result

__global__ void upsweep(int *output, int two_d)
{
    int two_dplus1 = 2 * two_d;
    int i = (blockIdx.x * blockDim.x + threadIdx.x) * two_dplus1;
    output[i + two_dplus1 - 1] += output[i + two_d - 1];
}

__global__ void downsweep(int *output, int two_d)
{
    int two_dplus1 = 2 * two_d;
    int i = (blockIdx.x * blockDim.x + threadIdx.x) * two_dplus1;
    int t = output[i + two_d - 1];
    output[i + two_d - 1] = output[i + two_dplus1 - 1];
    output[i + two_dplus1 - 1] += t;
}

using std::cout;
using std::endl;

void exclusive_scan(int *input, int N, int *result)
{
    const int blockSize = 256;

    hipMemcpy(result, input, N * sizeof(int), hipMemcpyDeviceToDevice);

    N = nextPow2(N);

    auto getSize = [&](int two_d) -> std::pair<int, int>
    {
        int two_dplus1 = 2 * two_d;
        // 计算总共需要多少个线程
        int threads_count = N / two_dplus1;

        int blocks_count = 1;
        // 如果超过了一个线程块的大小, 就拆出来
        if (threads_count > blockSize)
        {
            blocks_count = threads_count / blockSize;
            threads_count = blockSize;
        }

        return {blocks_count, threads_count};
    };
    
    for (int two_d = 1; two_d <= N / 2; two_d *= 2)
    {
        auto [blocks_count, threads_count] = getSize(two_d);
        upsweep<<<blocks_count, threads_count>>>(result, two_d);
    }

    int *tmp = (int *)malloc(sizeof(int));
    *tmp = 0;
    hipMemcpy(result + N - 1, tmp, sizeof(int), hipMemcpyHostToDevice);

    for (int two_d = N / 2; two_d >= 1; two_d /= 2)
    {
        auto [blocks_count, threads_count] = getSize(two_d);
        downsweep<<<blocks_count, threads_count>>>(result, two_d);
    }
}

//
// cudaScan --
//
// This function is a timing wrapper around the student's
// implementation of scan - it copies the input to the GPU
// and times the invocation of the exclusive_scan() function
// above. Students should not modify it.
double cudaScan(int *inarray, int *end, int *resultarray)
{
    int *device_result;
    int *device_input;
    int N = end - inarray;

    // This code rounds the arrays provided to exclusive_scan up
    // to a power of 2, but elements after the end of the original
    // input are left uninitialized and not checked for correctness.
    //
    // Student implementations of exclusive_scan may assume an array's
    // allocated length is a power of 2 for simplicity. This will
    // result in extra work on non-power-of-2 inputs, but it's worth
    // the simplicity of a power of two only solution.

    int rounded_length = nextPow2(end - inarray);

    hipMalloc((void **)&device_result, sizeof(int) * rounded_length);
    hipMalloc((void **)&device_input, sizeof(int) * rounded_length);

    // For convenience, both the input and output vectors on the
    // device are initialized to the input values. This means that
    // students are free to implement an in-place scan on the result
    // vector if desired.  If you do this, you will need to keep this
    // in mind when calling exclusive_scan from find_repeats.
    hipMemcpy(device_input, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_result, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_input, N, device_result);

    // Wait for completion
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    hipMemcpy(resultarray, device_result, (end - inarray) * sizeof(int), hipMemcpyDeviceToHost);

    double overallDuration = endTime - startTime;
    return overallDuration;
}

// cudaScanThrust --
//
// Wrapper around the Thrust library's exclusive scan function
// As above in cudaScan(), this function copies the input to the GPU
// and times only the execution of the scan itself.
//
// Students are not expected to produce implementations that achieve
// performance that is competition to the Thrust version, but it is fun to try.
double cudaScanThrust(int *inarray, int *end, int *resultarray)
{

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);

    hipMemcpy(d_input.get(), inarray, length * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    hipMemcpy(resultarray, d_output.get(), length * sizeof(int), hipMemcpyDeviceToHost);

    thrust::device_free(d_input);
    thrust::device_free(d_output);

    double overallDuration = endTime - startTime;
    return overallDuration;
}

// find_repeats --
//
// Given an array of integers `device_input`, returns an array of all
// indices `i` for which `device_input[i] == device_input[i+1]`.
//
// Returns the total number of pairs found

__global__ void compare(int *input, int length, int *output)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= length - 1)
        return;
    output[idx] = (input[idx] == input[idx + 1]);
}

__global__ void setIdx(int *input, int length, int *output)
{
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= length - 1)
        return;

    if (input[idx] + 1 == input[idx + 1])
        output[input[idx]] = idx;
}


int find_repeats(int *device_input, int length, int *device_output)
{

    auto getSize = [&]() -> std::pair<int, int>
    {
        const int block_size = 512;

        if (length < block_size)
            return {1, length};
        return {(length + block_size - 1) / block_size, block_size};
    };

    auto [blocks_count, threads_count] = getSize();

    compare<<<blocks_count, threads_count>>>(device_input, length, device_output);
    exclusive_scan(device_output, length, device_input);
    setIdx<<<blocks_count, threads_count>>>(device_input, length, device_output);


    int *res = (int *)malloc(sizeof(int));
    hipMemcpy(res, device_input + length - 1, sizeof(int), hipMemcpyDeviceToHost);

    return *res;
}

//
// cudaFindRepeats --
//
// Timing wrapper around find_repeats. You should not modify this function.
double cudaFindRepeats(int *input, int length, int *output, int *output_length)
{

    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);

    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    double startTime = CycleTimer::currentSeconds();

    int result = find_repeats(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    // set output count and results array
    *output_length = result;
    hipMemcpy(output, device_output, length * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    float duration = endTime - startTime;
    return duration;
}

void printCudaInfo()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i = 0; i < deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
}
