#include "hip/hip_runtime.h"
#include <string>
#include <algorithm>
#include <math.h>
#include <stdio.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "cudaRenderer.h"
#include "cycleTimer.h"
#include "image.h"
#include "noise.h"
#include "sceneLoader.h"
#include "util.h"

#define SCAN_BLOCK_DIM 1024
#define NUM_CIRCLES_PROC SCAN_BLOCK_DIM
#define TILE_WIDTH 64
#define TILE_HEIGHT 64
static short *cudaDeviceCircleFlag = nullptr;

////////////////////////////////////////////////////////////////////////////////////////
// Putting all the cuda kernels here
///////////////////////////////////////////////////////////////////////////////////////

struct GlobalConstants
{
    SceneName sceneName;

    int numCircles;
    int numCirclesProc; // No. of circles to process at a time.
    int numPixels;      // No. of pixels in the image.
    short numTilesX;
    short numTilesY;

    float *position;
    float *velocity;
    float *color;
    float *radius;

    int imageWidth;
    int imageHeight;
    float invWidth;
    float invHeight;
    short tileWidth;
    short tileHeight;

    float *imageData;
    short *circleFlag; // Flags indicating circles relevant to a tile or a pixel.
};

// Global variable that is in scope, but read-only, for all cuda
// kernels.  The __constant__ modifier designates this variable will
// be stored in special "constant" memory on the GPU. (we didn't talk
// about this type of memory in class, but constant memory is a fast
// place to put read-only variables).
__constant__ GlobalConstants cuConstRendererParams;

// read-only lookup tables used to quickly compute noise (needed by
// advanceAnimation for the snowflake scene)
__constant__ int cuConstNoiseYPermutationTable[256];
__constant__ int cuConstNoiseXPermutationTable[256];
__constant__ float cuConstNoise1DValueTable[256];

// color ramp table needed for the color ramp lookup shader
#define COLOR_MAP_SIZE 5
__constant__ float cuConstColorRamp[COLOR_MAP_SIZE][3];

// including parts of the CUDA code from external files to keep this
// file simpler and to seperate code that should not be modified
#include "circleBoxTest.cu_inl"
#include "exclusiveScan.cu_inl"
#include "noiseCuda.cu_inl"
#include "lookupColor.cu_inl"

// kernelClearImageSnowflake -- (CUDA device code)
//
// Clear the image, setting the image to the white-gray gradation that
// is used in the snowflake image
__global__ void kernelClearImageSnowflake()
{
    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float shade = .4f + .45f * static_cast<float>(height - imageY) / height;
    float4 value = make_float4(shade, shade, shade, 1.f);

    // write to global memory: As an optimization, I use a float4
    // store, that results in more efficient code than if I coded this
    // up as four seperate fp32 stores.
    *(float4 *)(&cuConstRendererParams.imageData[offset]) = value;
}

// kernelClearImage --  (CUDA device code)
//
// Clear the image, setting all pixels to the specified color rgba
__global__ void kernelClearImage(float r, float g, float b, float a)
{
    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float4 value = make_float4(r, g, b, a);

    // write to global memory: As an optimization, I use a float4
    // store, that results in more efficient code than if I coded this
    // up as four seperate fp32 stores.
    *(float4 *)(&cuConstRendererParams.imageData[offset]) = value;
}

// kernelAdvanceFireWorks
//
// Update the position of the fireworks (if circle is firework)
__global__ void kernelAdvanceFireWorks()
{
    const float dt = 1.f / 60.f;
    const float pi = 3.14159;
    const float maxDist = 0.25f;

    float *velocity = cuConstRendererParams.velocity;
    float *position = cuConstRendererParams.position;
    float *radius = cuConstRendererParams.radius;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= cuConstRendererParams.numCircles)
        return;

    if (0 <= index && index < NUM_FIREWORKS)
    { // firework center; no update
        return;
    }

    // determine the fire-work center/spark indices
    int fIdx = (index - NUM_FIREWORKS) / NUM_SPARKS;
    int sfIdx = (index - NUM_FIREWORKS) % NUM_SPARKS;

    int index3i = 3 * fIdx;
    int sIdx = NUM_FIREWORKS + fIdx * NUM_SPARKS + sfIdx;
    int index3j = 3 * sIdx;

    float cx = position[index3i];
    float cy = position[index3i + 1];

    // update position
    position[index3j] += velocity[index3j] * dt;
    position[index3j + 1] += velocity[index3j + 1] * dt;

    // fire-work sparks
    float sx = position[index3j];
    float sy = position[index3j + 1];

    // compute vector from firework-spark
    float cxsx = sx - cx;
    float cysy = sy - cy;

    // compute distance from fire-work
    float dist = sqrt(cxsx * cxsx + cysy * cysy);
    if (dist > maxDist)
    { // restore to starting position
        // random starting position on fire-work's rim
        float angle = (sfIdx * 2 * pi) / NUM_SPARKS;
        float sinA = sin(angle);
        float cosA = cos(angle);
        float x = cosA * radius[fIdx];
        float y = sinA * radius[fIdx];

        position[index3j] = position[index3i] + x;
        position[index3j + 1] = position[index3i + 1] + y;
        position[index3j + 2] = 0.0f;

        // travel scaled unit length
        velocity[index3j] = cosA / 5.0;
        velocity[index3j + 1] = sinA / 5.0;
        velocity[index3j + 2] = 0.0f;
    }
}

// kernelAdvanceHypnosis
//
// Update the radius/color of the circles
__global__ void kernelAdvanceHypnosis()
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= cuConstRendererParams.numCircles)
        return;

    float *radius = cuConstRendererParams.radius;

    float cutOff = 0.5f;
    // place circle back in center after reaching threshold radisus
    if (radius[index] > cutOff)
    {
        radius[index] = 0.02f;
    }
    else
    {
        radius[index] += 0.01f;
    }
}

// kernelAdvanceBouncingBalls
//
// Update the positino of the balls
__global__ void kernelAdvanceBouncingBalls()
{
    const float dt = 1.f / 60.f;
    const float kGravity = -2.8f; // sorry Newton
    const float kDragCoeff = -0.8f;
    const float epsilon = 0.001f;

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= cuConstRendererParams.numCircles)
        return;

    float *velocity = cuConstRendererParams.velocity;
    float *position = cuConstRendererParams.position;

    int index3 = 3 * index;
    // reverse velocity if center position < 0
    float oldVelocity = velocity[index3 + 1];
    float oldPosition = position[index3 + 1];

    if (oldVelocity == 0.f && oldPosition == 0.f)
    { // stop-condition
        return;
    }

    if (position[index3 + 1] < 0 && oldVelocity < 0.f)
    { // bounce ball
        velocity[index3 + 1] *= kDragCoeff;
    }

    // update velocity: v = u + at (only along y-axis)
    velocity[index3 + 1] += kGravity * dt;

    // update positions (only along y-axis)
    position[index3 + 1] += velocity[index3 + 1] * dt;

    if (fabsf(velocity[index3 + 1] - oldVelocity) < epsilon && oldPosition < 0.0f && fabsf(position[index3 + 1] - oldPosition) < epsilon)
    { // stop ball
        velocity[index3 + 1] = 0.f;
        position[index3 + 1] = 0.f;
    }
}

// kernelAdvanceSnowflake -- (CUDA device code)
//
// move the snowflake animation forward one time step.  Updates circle
// positions and velocities.  Note how the position of the snowflake
// is reset if it moves off the left, right, or bottom of the screen.
__global__ void kernelAdvanceSnowflake()
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= cuConstRendererParams.numCircles)
        return;

    const float dt = 1.f / 60.f;
    const float kGravity = -1.8f; // sorry Newton
    const float kDragCoeff = 2.f;

    int index3 = 3 * index;

    float *positionPtr = &cuConstRendererParams.position[index3];
    float *velocityPtr = &cuConstRendererParams.velocity[index3];

    // loads from global memory
    float3 position = *((float3 *)positionPtr);
    float3 velocity = *((float3 *)velocityPtr);

    // hack to make farther circles move more slowly, giving the
    // illusion of parallax
    float forceScaling = fmin(fmax(1.f - position.z, .1f), 1.f); // clamp

    // add some noise to the motion to make the snow flutter
    float3 noiseInput;
    noiseInput.x = 10.f * position.x;
    noiseInput.y = 10.f * position.y;
    noiseInput.z = 255.f * position.z;
    float2 noiseForce = cudaVec2CellNoise(noiseInput, index);
    noiseForce.x *= 7.5f;
    noiseForce.y *= 5.f;

    // drag
    float2 dragForce;
    dragForce.x = -1.f * kDragCoeff * velocity.x;
    dragForce.y = -1.f * kDragCoeff * velocity.y;

    // update positions
    position.x += velocity.x * dt;
    position.y += velocity.y * dt;

    // update velocities
    velocity.x += forceScaling * (noiseForce.x + dragForce.y) * dt;
    velocity.y += forceScaling * (kGravity + noiseForce.y + dragForce.y) * dt;

    float radius = cuConstRendererParams.radius[index];

    // if the snowflake has moved off the left, right or bottom of
    // the screen, place it back at the top and give it a
    // pseudorandom x position and velocity.
    if ((position.y + radius < 0.f) ||
        (position.x + radius) < -0.f ||
        (position.x - radius) > 1.f)
    {
        noiseInput.x = 255.f * position.x;
        noiseInput.y = 255.f * position.y;
        noiseInput.z = 255.f * position.z;
        noiseForce = cudaVec2CellNoise(noiseInput, index);

        position.x = .5f + .5f * noiseForce.x;
        position.y = 1.35f + radius;

        // restart from 0 vertical velocity.  Choose a
        // pseudo-random horizontal velocity.
        velocity.x = 2.f * noiseForce.y;
        velocity.y = 0.f;
    }

    // store updated positions and velocities to global memory
    *((float3 *)positionPtr) = position;
    *((float3 *)velocityPtr) = velocity;
}

__device__ __inline__ bool
shouldRender(int circleIndex, float2 &pixelCenterNorm, float *pixelDist)
{
    int index3 = 3 * circleIndex;

    // read position and radius
    float3 p = *(float3 *)(&cuConstRendererParams.position[index3]);
    float rad = cuConstRendererParams.radius[circleIndex];
    float maxDist = rad * rad;

    float diffX = p.x - pixelCenterNorm.x;
    float diffY = p.y - pixelCenterNorm.y;
    *pixelDist = diffX * diffX + diffY * diffY;
    return *pixelDist <= maxDist;
}

// Applies the changes induced by a batch of circles for a pixel.
__global__ void applyShadePixel(int offset, int numTrueCircles)
{
    int pixelIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (pixelIndex >= cuConstRendererParams.numPixels)
    {
        return;
    }

    short imageWidth = cuConstRendererParams.imageWidth;
    short pixelY = pixelIndex / imageWidth;
    short pixelX = pixelIndex % imageWidth;
    float invWidth = cuConstRendererParams.invWidth;
    float invHeight = cuConstRendererParams.invHeight;
    float2 pixelCenterNorm = make_float2(invWidth * (static_cast<float>(pixelX) + 0.5f),
                                         invHeight * (static_cast<float>(pixelY) + 0.5f));

    short tileWidth = cuConstRendererParams.tileWidth;
    short tileHeight = cuConstRendererParams.tileHeight;
    short numTilesX = cuConstRendererParams.numTilesX;
    short tileIndex = (pixelY / tileHeight) * numTilesX + (pixelX / tileWidth);

    // BEGIN SHOULD-BE-ATOMIC REGION
    float4 *imagePtr = (float4 *)(&cuConstRendererParams.imageData[4 * pixelIndex]);
    float4 newColor = *imagePtr;
    const int numCirclesProc = cuConstRendererParams.numCirclesProc;
    if (cuConstRendererParams.sceneName == SNOWFLAKES ||
        cuConstRendererParams.sceneName == SNOWFLAKES_SINGLE_FRAME)
    {
        const float kCircleMaxAlpha = .5f;
        const float falloffScale = 4.f;

        for (int idx = 0; idx < numTrueCircles; ++idx)
        {
            const int flagIndex = numCirclesProc * tileIndex + idx;
            const short flagValue = cuConstRendererParams.circleFlag[flagIndex];
            if (flagValue < 0)
            {
                break;
            }

            const int circleIndex = offset + flagValue;
            float pixelDist;
            if (!shouldRender(circleIndex, pixelCenterNorm, &pixelDist))
            {
                continue;
            }

            // Compute the RGBA of the circle for this pixel.
            float3 rgb;
            float alpha;

            int index3 = 3 * circleIndex;
            float p_z = *(float *)(&cuConstRendererParams.position[index3 + 2]);
            float rad = cuConstRendererParams.radius[circleIndex];
            float normPixelDist = sqrt(pixelDist) / rad;
            rgb = lookupColor(normPixelDist);

            float maxAlpha = .6f + .4f * (1.f - p_z);
            maxAlpha = kCircleMaxAlpha * fmaxf(fminf(maxAlpha, 1.f), 0.f); // kCircleMaxAlpha * clamped value
            alpha = maxAlpha * exp(-1.f * falloffScale * normPixelDist * normPixelDist);

            float oneMinusAlpha = 1.f - alpha;

            newColor.x = alpha * rgb.x + oneMinusAlpha * newColor.x;
            newColor.y = alpha * rgb.y + oneMinusAlpha * newColor.y;
            newColor.z = alpha * rgb.z + oneMinusAlpha * newColor.z;
            newColor.w = alpha + newColor.w;
        }
    }
    else
    {
        float alpha = .5f;
        float oneMinusAlpha = 1.f - alpha;
        for (int idx = 0; idx < numTrueCircles; ++idx)
        {
            const int flagIndex = numCirclesProc * tileIndex + idx;
            const short flagValue = cuConstRendererParams.circleFlag[flagIndex];
            if (flagValue < 0)
            {
                break;
            }

            const int circleIndex = offset + flagValue;
            float pixelDist;
            if (!shouldRender(circleIndex, pixelCenterNorm, &pixelDist))
            {
                continue;
            }

            // Compute the RGBA of the circle for this pixel.
            int index3 = 3 * circleIndex;
            float3 rgb = *(float3 *)(&cuConstRendererParams.color[index3]);

            newColor.x = alpha * rgb.x + oneMinusAlpha * newColor.x;
            newColor.y = alpha * rgb.y + oneMinusAlpha * newColor.y;
            newColor.z = alpha * rgb.z + oneMinusAlpha * newColor.z;
            newColor.w = alpha + newColor.w;
        }
    }
    *imagePtr = newColor;
    // END SHOULD-BE-ATOMIC REGION
}

__device__ __inline__ short isCircleInBox(int circleIndex, short tileX, short tileY)
{
    if (circleIndex >= cuConstRendererParams.numCircles)
    {
        return 0;
    }

    short tileWidth = cuConstRendererParams.tileWidth;
    short tileHeight = cuConstRendererParams.tileHeight;
    float invWidth = cuConstRendererParams.invWidth;
    float invHeight = cuConstRendererParams.invHeight;

    short boxL = tileWidth * tileX;
    short boxR = boxL + tileWidth;
    short boxB = tileHeight * tileY;
    short boxT = boxB + tileHeight;
    float nBoxL = invWidth * static_cast<float>(boxL);
    float nBoxR = invWidth * static_cast<float>(boxR);
    float nBoxB = invHeight * static_cast<float>(boxB);
    float nBoxT = invHeight * static_cast<float>(boxT);

    int index3 = 3 * circleIndex;
    float3 p = *(float3 *)(&cuConstRendererParams.position[index3]);
    float rad = cuConstRendererParams.radius[circleIndex];

    short circleFlag = circleInBox(p.x, p.y, rad, nBoxL, nBoxR, nBoxT, nBoxB);
    return circleFlag;
}
#include <assert.h>

__global__ void setTileCircles(int offset, int numTrueCircles)
{
    short circleIdx = threadIdx.x;
    const int circleIndex = offset + circleIdx;

    __shared__ uint prefixSumInput[SCAN_BLOCK_DIM];
    prefixSumInput[circleIdx] = isCircleInBox(circleIndex, blockIdx.x, blockIdx.y);

    __syncthreads(); // Wait until all circles for the tile finish.

    __shared__ uint prefixSumOutput[SCAN_BLOCK_DIM];
    __shared__ uint prefixSumScratch[2 * SCAN_BLOCK_DIM];
    sharedMemExclusiveScan(circleIdx, prefixSumInput, prefixSumOutput,
                           prefixSumScratch, SCAN_BLOCK_DIM);
    // Extra threads are only used for the scan.
    if (circleIndex >= cuConstRendererParams.numCircles)
        return;

    short tileIndex = blockIdx.y * cuConstRendererParams.numTilesX + blockIdx.x;

    const int numCirclesProc = cuConstRendererParams.numCirclesProc;
    int flagIndex = numCirclesProc * tileIndex + prefixSumOutput[circleIdx];
    
    if (circleIdx < numTrueCircles - 1 &&
        prefixSumOutput[circleIdx] != prefixSumOutput[circleIdx + 1])
    {
        cuConstRendererParams.circleFlag[flagIndex] = circleIdx;
    }
    else if (circleIdx == numTrueCircles - 1)
    {
        if (prefixSumInput[circleIdx] == 1)
            cuConstRendererParams.circleFlag[flagIndex] = circleIdx;
            
        int endIndex = flagIndex + prefixSumInput[circleIdx];
        if (endIndex < numCirclesProc * (tileIndex + 1))
            cuConstRendererParams.circleFlag[endIndex] = -1;
    }
}

////////////////////////////////////////////////////////////////////////////////////////

CudaRenderer::CudaRenderer()
{
    image = NULL;

    numCircles = 0;

    position = NULL;
    velocity = NULL;
    color = NULL;
    radius = NULL;

    cudaDevicePosition = NULL;
    cudaDeviceVelocity = NULL;
    cudaDeviceColor = NULL;
    cudaDeviceRadius = NULL;
    cudaDeviceImageData = NULL;
}

CudaRenderer::~CudaRenderer()
{
    if (image)
    {
        delete image;
    }

    if (position)
    {
        delete[] position;
        delete[] velocity;
        delete[] color;
        delete[] radius;
    }

    if (cudaDevicePosition)
    {
        hipFree(cudaDevicePosition);
        hipFree(cudaDeviceVelocity);
        hipFree(cudaDeviceColor);
        hipFree(cudaDeviceRadius);
        hipFree(cudaDeviceImageData);
        hipFree(cudaDeviceCircleFlag);
    }
}

const Image *CudaRenderer::getImage()
{
    // need to copy contents of the rendered image from device memory
    // before we expose the Image object to the caller
    printf("Copying image data from device\n");

    hipMemcpy(image->data,
               cudaDeviceImageData,
               sizeof(float) * 4 * image->width * image->height,
               hipMemcpyDeviceToHost);

    return image;
}

void CudaRenderer::loadScene(SceneName scene)
{
    sceneName = scene;
    loadCircleScene(sceneName, numCircles, position, velocity, color, radius);
}

void CudaRenderer::setup()
{
    int deviceCount = 0;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for CudaRenderer\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i = 0; i < deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");

    // By this time the scene should be loaded.  Now copy all the key
    // data structures into device memory so they are accessible to
    // CUDA kernels
    //
    // See the CUDA Programmer's Guide for descriptions of
    // hipMalloc and hipMemcpy

    hipMalloc(&cudaDevicePosition, sizeof(float) * 3 * numCircles);
    hipMalloc(&cudaDeviceVelocity, sizeof(float) * 3 * numCircles);
    hipMalloc(&cudaDeviceColor, sizeof(float) * 3 * numCircles);
    hipMalloc(&cudaDeviceRadius, sizeof(float) * numCircles);
    hipMalloc(&cudaDeviceImageData, sizeof(float) * 4 * image->width * image->height);
    short numTilesX = (image->width + TILE_WIDTH - 1) / TILE_WIDTH;
    short numTilesY = (image->height + TILE_HEIGHT - 1) / TILE_HEIGHT;
    hipMalloc(&cudaDeviceCircleFlag, sizeof(short) * NUM_CIRCLES_PROC * numTilesX * numTilesY);

    hipMemcpy(cudaDevicePosition, position, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceVelocity, velocity, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceColor, color, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceRadius, radius, sizeof(float) * numCircles, hipMemcpyHostToDevice);

    // Initialize parameters in constant memory.  We didn't talk about
    // constant memory in class, but the use of read-only constant
    // memory here is an optimization over just sticking these values
    // in device global memory.  NVIDIA GPUs have a few special tricks
    // for optimizing access to constant memory.  Using global memory
    // here would have worked just as well.  See the Programmer's
    // Guide for more information about constant memory.

    GlobalConstants params;
    params.sceneName = sceneName;
    params.numCircles = numCircles;
    params.numCirclesProc = NUM_CIRCLES_PROC;
    params.numPixels = image->width * image->height;
    params.numTilesX = numTilesX;
    params.numTilesY = numTilesY;
    params.tileWidth = TILE_WIDTH;
    params.tileHeight = TILE_HEIGHT;
    params.imageWidth = image->width;
    params.imageHeight = image->height;
    params.invWidth = 1.f / image->width;
    params.invHeight = 1.f / image->height;
    params.position = cudaDevicePosition;
    params.velocity = cudaDeviceVelocity;
    params.color = cudaDeviceColor;
    params.radius = cudaDeviceRadius;
    params.imageData = cudaDeviceImageData;
    params.circleFlag = cudaDeviceCircleFlag;

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstRendererParams), &params, sizeof(GlobalConstants));

    // also need to copy over the noise lookup tables, so we can
    // implement noise on the GPU
    int *permX;
    int *permY;
    float *value1D;
    getNoiseTables(&permX, &permY, &value1D);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseXPermutationTable), permX, sizeof(int) * 256);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseYPermutationTable), permY, sizeof(int) * 256);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoise1DValueTable), value1D, sizeof(float) * 256);

    // last, copy over the color table that's used by the shading
    // function for circles in the snowflake demo

    float lookupTable[COLOR_MAP_SIZE][3] = {
        {1.f, 1.f, 1.f},
        {1.f, 1.f, 1.f},
        {.8f, .9f, 1.f},
        {.8f, .9f, 1.f},
        {.8f, 0.8f, 1.f},
    };

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstColorRamp), lookupTable, sizeof(float) * 3 * COLOR_MAP_SIZE);
}

// allocOutputImage --
//
// Allocate buffer the renderer will render into.  Check status of
// image first to avoid memory leak.
void CudaRenderer::allocOutputImage(int width, int height)
{
    if (image)
        delete image;
    image = new Image(width, height);
}

// clearImage --
//
// Clear's the renderer's target image.  The state of the image after
// the clear depends on the scene being rendered.
void CudaRenderer::clearImage()
{
    // 256 threads per block is a healthy number
    dim3 blockDim(16, 16, 1);
    dim3 gridDim((image->width + blockDim.x - 1) / blockDim.x,
                 (image->height + blockDim.y - 1) / blockDim.y);

    if (sceneName == SNOWFLAKES || sceneName == SNOWFLAKES_SINGLE_FRAME)
    {
        kernelClearImageSnowflake<<<gridDim, blockDim>>>();
    }
    else
    {
        kernelClearImage<<<gridDim, blockDim>>>(1.f, 1.f, 1.f, 1.f);
    }
    hipDeviceSynchronize();
}

// advanceAnimation --
//
// Advance the simulation one time step.  Updates all circle positions
// and velocities
void CudaRenderer::advanceAnimation()
{
    // 256 threads per block is a healthy number
    dim3 blockDim(256, 1);
    dim3 gridDim((numCircles + blockDim.x - 1) / blockDim.x);

    // only the snowflake scene has animation
    if (sceneName == SNOWFLAKES)
    {
        kernelAdvanceSnowflake<<<gridDim, blockDim>>>();
    }
    else if (sceneName == BOUNCING_BALLS)
    {
        kernelAdvanceBouncingBalls<<<gridDim, blockDim>>>();
    }
    else if (sceneName == HYPNOSIS)
    {
        kernelAdvanceHypnosis<<<gridDim, blockDim>>>();
    }
    else if (sceneName == FIREWORKS)
    {
        kernelAdvanceFireWorks<<<gridDim, blockDim>>>();
    }
    hipDeviceSynchronize();
}

void CudaRenderer::render()
{
    // 256 threads per block is a healthy number
    dim3 blockDim(256, 1);
    dim3 pixelDim((image->width * image->height + blockDim.x - 1) / blockDim.x);

    short numTilesX = (image->width + TILE_WIDTH - 1) / TILE_WIDTH;
    short numTilesY = (image->height + TILE_HEIGHT - 1) / TILE_HEIGHT;
    dim3 tileDim(numTilesX, numTilesY);

    for (int offset = 0; offset < numCircles; offset += NUM_CIRCLES_PROC)
    {
        const int numTrueCircles = std::min(NUM_CIRCLES_PROC, numCircles - offset);
        setTileCircles<<<tileDim, NUM_CIRCLES_PROC>>>(offset, numTrueCircles);
        applyShadePixel<<<pixelDim, blockDim>>>(offset, numTrueCircles);
        hipDeviceSynchronize();
    }
}